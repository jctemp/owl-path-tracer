#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#

__global__ void precision_test(float* dest)
{
    dest[0] = .3f + .5f;
}

// Demonstrate some basic assertions.
TEST(HelloTest, BasicAssertions) {

    
    float* host = (float *)malloc(sizeof(float));
    float* device = nullptr;
    hipMalloc(&device, sizeof(float));
    hipMemcpy(device, host, sizeof(float), hipMemcpyHostToDevice);

    precision_test<<<1, 1>>>(device);

    hipMemcpy(host, device, sizeof(float), hipMemcpyDeviceToHost);

    printf("%f\n", host[0]);

    free(host);
    hipFree(device);

    // Expect two strings not to be equal.
    EXPECT_STRNE("hello", "world");
    // Expect equality.
    EXPECT_EQ(7 * 6, 42);
}