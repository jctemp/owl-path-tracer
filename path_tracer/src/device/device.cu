#include "hip/hip_runtime.h"
﻿#include <owl/owl_device.h>

#include "types.hpp"
#include "device.hpp"
#include "materials.hpp"
#include "macros.hpp"

using radiance_ray = owl::RayT<0, 2>;
using shadow_ray = owl::RayT<1, 2>;

/// this constant must be called optixLaunchParams<br>
/// it is declared in path_tracer extern/owl/owl/DeviceContext.cpp
__constant__ launch_params_data optixLaunchParams;


inline __device__ vec3 pow(vec3 const& v, float const& p)
{
    return vec3{powf(v.x, p), powf(v.y, p), powf(v.z, p)};
}


inline __device__ vec2 uv_on_sphere(vec3 n)
{
    auto const u{0.5f + atan2(n.x, n.z) / (2.0f * pi)};
    auto const v{0.5f + asin(n.y) / pi};
    return vec2{u, v};
}


inline __device__ vec3 sample_environment(vec3 dir)
{
    auto& launch_params {optixLaunchParams};

    vec2 tc{uv_on_sphere(dir)};
    owl::vec4f const texColor{
            tex2D<float4>(launch_params.environment_map, tc.x, tc.y)};
    return vec3{texColor};
}


inline __device__ void load_triangle_indices(int32_t const& mesh_index, int32_t const& primitive_id,
                                             ivec3& indices)
{
    auto& launch_params = optixLaunchParams;
    get_data(auto indices_buffer, launch_params.indices_buffer, mesh_index, Buffer);
    get_data(indices, indices_buffer, primitive_id, ivec3);
}

inline __device__ void load_triangle_vertices(int32_t const& mesh_index, ivec3 const& indices, vec2 const& barycentric,
                                       vec3& position, vec3& geometric_normal)
{
    auto& launch_params = optixLaunchParams;
    get_data(auto vertices_buffer, launch_params.vertices_buffer, mesh_index, Buffer);
    get_data(auto p0, vertices_buffer, indices.x, vec3);
    get_data(auto p1, vertices_buffer, indices.y, vec3);
    get_data(auto p2, vertices_buffer, indices.z, vec3);

    position = (1 - barycentric.x - barycentric.y) * p0 + barycentric.x * p1 + barycentric.y * p2;
    geometric_normal = normalize(cross(p1 - p0, p2 - p0));
}

inline __device__ void load_triangle_normals(int32_t const& mesh_index, ivec3 const& indices, vec2 const& barycentric,
                                      vec3& shading_normal)
{
    auto& launch_params = optixLaunchParams;
    get_data(auto normals_buffer, launch_params.normals_buffer, mesh_index, Buffer);
    get_data(auto n0, normals_buffer, indices.x, vec3);
    get_data(auto n1, normals_buffer, indices.y, vec3);
    get_data(auto n2, normals_buffer, indices.z, vec3);

    shading_normal = normalize((1 - barycentric.x - barycentric.y) * n0 + barycentric.x * n1 + barycentric.y * n2);
}


inline __device__ bool visibiliy_test(vec3 const& position, vec3 const& direction, float const& max_distance)
{
    auto& launch_params {optixLaunchParams};

    // check the space between position and target only
    shadow_ray light_ray{position, direction, t_min, max_distance - t_min};
    bool visible{false};

    owl::traceRay(launch_params.world, light_ray, visible,
              OPTIX_RAY_FLAG_DISABLE_ANYHIT
            | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT);

    return visible;
}


__device__ vec3 trace_path(radiance_ray& ray, random& random, int32_t& samples)
{
    auto& launch_params = optixLaunchParams;

    /// total accumulation of existent radiance
    vec3 radiance{0.0f};


    /// keeps tracks of the throughput of a ray, it is weighting the radiance <br>
    /// beta = f * cos(theta) / pdf
    vec3 beta{1.0f};

    hit_data hd;
    material_data ms;
    per_ray_data prd{random, scatter_event::none, &hd, &ms};
    material_type sampled_type{};

    for (int32_t depth{0}; depth < launch_params.max_path_depth; ++depth)
    {
        /// find closest intersection
        owl::traceRay(launch_params.world, ray, prd);

        /// miss then terminate the path and sample environment
        if (prd.scatter_event == scatter_event::miss)
        {
            if (launch_params.environment_use && launch_params.environment_map)
                radiance += sample_environment(ray.direction) * beta;
            else if (launch_params.environment_auto)
                radiance += lerp(vec3{1.0f}, vec3{0.5f, 0.7f, 1.0f}, 0.5f * (ray.direction.y + 1.0f)) * beta;
            else
                radiance += launch_params.environment_color * launch_params.environment_intensity * beta;
            break;
        }

        /// load mesh for interaction calculations
        ivec3 indices{};
        vec3 v_p{}, v_gn{}, v_n{};

        load_triangle_indices(hd.mesh_index, hd.primitive_index, indices);
        load_triangle_vertices(hd.mesh_index, indices, hd.barycentric, v_p, v_gn);
        load_triangle_normals(hd.mesh_index, indices, hd.barycentric, v_n);

        vec3 wo{hd.wo}, wi{};

        vec3 T{}, B{};
        onb(v_n, T, B);

        float pdf{};
        vec3 f{};

        material_data material{};
        if (hd.material_index >= 0)
        {
            get_data(material, launch_params.material_buffer, hd.material_index, material_data);
        }

        vec3 local_wo{to_local(T, B, v_n, wo)}, local_wi{}, local_wh{};

        sample_lambert(material, local_wo, prd.random, local_wi, f, pdf, sampled_type);

        //sample_disney_bsdf(material, local_wo, prd.random,
        //        local_wi, local_wh, f, pdf, sampled_type);

        wi = to_world(T, B, v_n, local_wi);

        /// terminate or catching de-generate paths
        if (pdf < 1E-5f)
            break;

        if (has_inf(f) || has_nan(f))
        {
            --depth; // invalid path and re-sample
            continue;
        }

        beta *= (f * owl::abs(owl::dot(wi, v_n))) / pdf;

        /// terminate path by random
        auto const beta_max{owl::max(beta.x, owl::max(beta.y, beta.z))};
        if (depth > 3)
        {
            float q{owl::max(.05f, 1 - beta_max)};
            if (prd.random() > q) break;
        }

        ray = radiance_ray{v_p, wi, t_min, t_max};
    }

    return radiance;
}

OPTIX_RAYGEN_PROGRAM(ray_gen)()
{
    auto& launch_params = optixLaunchParams;

    ray_gen_data const& self{owl::getProgramData<ray_gen_data>()};
    ivec2 const pixelId{owl::getLaunchIndex()};
    random pxRand{(uint32_t) pixelId.x, (uint32_t) pixelId.y};

    vec3 color{0.0f};
    for (int32_t s{0}; s < launch_params.max_samples; ++s)
    {
        // shot ray with slight randomness to make soft edges
        vec2 const rand{pxRand(), pxRand()};
        vec2 const screen{(vec2{pixelId} + rand) / vec2{self.fb_size}};

        // determine initial ray form the camera
        radiance_ray ray{
                self.camera.origin,
                normalize(
                        self.camera.llc + screen.u * self.camera.horizontal + screen.v * self.camera.vertical -
                        self.camera.origin),
                t_min, t_max};

        color += trace_path(ray, pxRand, s);
    }

    // take the average of all samples per pixel and apply gamma correction
    color *= 1.0f / static_cast<float>(launch_params.max_samples);
    color = o_saturate(pow(color, 1.0f / 2.2f));

    // save result into the buffer
    const int fbOfs = pixelId.x + self.fb_size.x * (self.fb_size.y - 1 - pixelId.y);
    self.fb_ptr[fbOfs]
            = owl::make_rgba(color);
}

OPTIX_CLOSEST_HIT_PROGRAM(triangle_hit)()
{
    per_ray_data& prd{owl::getPRD<per_ray_data>()};

    auto const& barycentric{optixGetTriangleBarycentrics()};
    auto const& ray_t_max{optixGetRayTmax()};
    auto const& ray_direction{optixGetWorldRayDirection()};
    auto const& primitive_index{optixGetPrimitiveIndex()};

    prd.hd->barycentric = barycentric;
    prd.hd->t = ray_t_max;
    prd.hd->wo = ray_direction;
    prd.hd->wo = -normalize(prd.hd->wo);
    prd.hd->primitive_index = primitive_index;

    auto const& self = owl::getProgramData<entity_data>();

    prd.hd->material_index = self.material_index;
    prd.hd->mesh_index = self.mesh_index;

    prd.scatter_event = scatter_event::hit;
}

OPTIX_MISS_PROGRAM(miss)()
{
    per_ray_data& prd{owl::getPRD<per_ray_data>()};
    prd.scatter_event = scatter_event::miss;
}

OPTIX_MISS_PROGRAM(miss_shadow)()
{
    bool& prd{owl::getPRD<bool>()};
    prd = true;
}