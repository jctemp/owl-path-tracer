#include "hip/hip_runtime.h"
﻿
#include "device.hpp"
#include "core/core.hpp"

PT_DEVICE_CONSTANT LaunchParams optixLaunchParams;

OPTIX_RAYGEN_PROGRAM(rayGenenration)()
{
	RayGenData const& self{ owl::getProgramData<RayGenData>() };
	Int2 const pixelId{ owl::getLaunchIndex() };
	Random pxRand{ (Uint)pixelId.x, (Uint)pixelId.y };

	Float3 color{ 0.0f };
	for (Int s{ 0 }; s < optixLaunchParams.samplesPerPixel; ++s)
	{
		// shot ray with slight randomness to make soft edges
		Float2 const rand{ pxRand(), pxRand() };
		Float2 const screen{ (Float2{pixelId} + rand) / Float2{self.fbSize} };

		// determine initial ray form the camera
		owl::Ray ray{ self.camera.origin, owl::normalize(self.camera.llc
			+ screen.u * self.camera.horizontal
			+ screen.v * self.camera.vertical
			- self.camera.origin), T_MIN, T_MAX };

		color += tracePath(ray, pxRand);
	}

	// take the average of all samples per pixel and apply gamma correction
	color *= 1.0f / optixLaunchParams.samplesPerPixel;
	color = owl::sqrt(color);
	color = saturate<Float3>(color);


	// save result into the buffer
	const int fbOfs = pixelId.x + self.fbSize.x * (self.fbSize.y - 1 - pixelId.y);
	self.fbPtr[fbOfs]
		= owl::make_rgba(color);
}

